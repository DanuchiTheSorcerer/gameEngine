#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <windows.h>
#include <chrono>
#include <climits>
#include "winlib.h"

int width = 1000;
int height = 1000;
float targetTPS = 60;
int refreshRate;
BITMAPINFO bmi;
int currentActiveInterpolator = 0; //cpu side
hipEvent_t interpolatorCopyEvent;
hipEvent_t fpsCopyEvent;
int frameCounter;

struct interpolator {
    //fully customisable by the person using the engine. Contains all the information needed to compute a frame, minus the interpolation index
    int tickCount; // example data
};

struct gpuMeta {
    uint32_t* frame; //array of buffers
    interpolator interpolators[2]; // array of interpolators
    int activeInterpolator = 0; // index of the current interpolator being used
    uint32_t* pointerToDisplay;
    int framesCalculated;

    //flags, set only by the CPU and interpreted by GPU when it has the chance
    bool shouldSwitchInterpolator = false;  // for switching interpolator, set after new one has been copied
    bool shouldEndKernel = false; // for ending the persistent kernel
    bool shouldCopyFrame = false;  // for copying to mapped memory
};

gpuMeta* gpuMetaData; // data on the GPU
uint32_t* displayFrame; //pined mapped memory buffer on DRAM for displaying

void initAll(HINSTANCE hInstance) {
    //intiate winlib
    WinLib_Init(hInstance);

    //get refresh rate
    DEVMODE dm = {0};
    dm.dmSize = sizeof(DEVMODE);
    EnumDisplaySettings(nullptr, ENUM_CURRENT_SETTINGS, &dm);
    refreshRate = dm.dmDisplayFrequency;

    //configure bit map
    ZeroMemory(&bmi, sizeof(bmi));
    bmi.bmiHeader.biSize = sizeof(BITMAPINFOHEADER);
    bmi.bmiHeader.biWidth = width;
    bmi.bmiHeader.biHeight = -height;
    bmi.bmiHeader.biPlanes = 1;
    bmi.bmiHeader.biBitCount = 32;
    bmi.bmiHeader.biCompression = BI_RGB;

    //cuda memory allocations
    hipMalloc(&gpuMetaData,sizeof(gpuMeta));   // allocate GPU data
    gpuMeta initialData = {};  

    hipMalloc(&initialData.frame, width * height * sizeof(uint32_t));  //allocate frame
    hipHostAlloc(&displayFrame, width * height * sizeof(uint32_t), hipHostMallocMapped); //allocate display frame to pinned mapped memory for quick copies by the GPU

    uint32_t* deviceDisplayPtr = nullptr;
    hipHostGetDevicePointer(&deviceDisplayPtr, displayFrame, 0);
    initialData.pointerToDisplay = deviceDisplayPtr; // copy pinter so that the GPU can copy frames to CPU mapped memory

    hipMemcpy(gpuMetaData,&initialData,sizeof(gpuMeta), hipMemcpyHostToDevice);

    hipEventCreate(&interpolatorCopyEvent);
    hipEventCreate(&fpsCopyEvent);
}

void PaintWindow(HDC hdc) {
    //draw (this is called by winlib when handling an invalidated rect)
    StretchDIBits(hdc, 0, 0, width, height, 0, 0, width, height,displayFrame, &bmi,DIB_RGB_COLORS, SRCCOPY);
}

__global__ void computeFrame(uint32_t* buffer, interpolator* interpolator,int width,int height) { //pointer to buffer
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    const int idx = y * width + x;

    buffer[idx] = 0xFF000000 | 
        ((x + interpolator->tickCount*5) % 255) << 16 |
        ((y + interpolator->tickCount*5) % 255) << 8 |
        ((x + y + interpolator->tickCount*5) % 255);
}

__global__ void copyFrameKernel(uint32_t* dst, const uint32_t* src, int totalPixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalPixels) {
        dst[idx] = src[idx];
    }
}

__global__ void frameComputeLoop(gpuMeta* gpuMetaData, int width, int height,hipStream_t stream) {
    while(!gpuMetaData->shouldEndKernel) {
        //check for new interpolators

        if (gpuMetaData->shouldSwitchInterpolator) {
            gpuMetaData->activeInterpolator = 1 - gpuMetaData->activeInterpolator;
            gpuMetaData->shouldSwitchInterpolator = false;
        }

        // compute a buffer
        dim3 threadsPerBlock(16, 16);     
        dim3 numBlocks((width + 15) / 16, (height + 15) / 16);



        // Launch the frame computation
        computeFrame<<<numBlocks, threadsPerBlock>>>(gpuMetaData->frame,&gpuMetaData->interpolators[gpuMetaData->activeInterpolator],width, height);

        gpuMetaData->framesCalculated = gpuMetaData->framesCalculated + 1;
        //copy frame if needed
        if (gpuMetaData->shouldCopyFrame) {
            int totalPixels = width * height;
            int threads = 256;
            int blocks = (totalPixels + threads - 1) / threads;
            copyFrameKernel<<<blocks, threads>>>(gpuMetaData->pointerToDisplay, gpuMetaData->frame, totalPixels);
        
            // Optionally, use __threadfence() to ensure memory visibility if needed.
            __threadfence();
        
            // Once copy is complete, reset the flag.
            gpuMetaData->shouldCopyFrame = false;
        }


    }
}

int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpCmdLine, int nCmdShow) {
    initAll(hInstance); //initialize
    WinWindow* window = WinLib_CreateWindow("CUDA Powered Engine", width, height, hInstance); //open the window

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);



    auto lastTickTime = std::chrono::high_resolution_clock::now();
    auto lastDisplayTime = std::chrono::high_resolution_clock::now();
    int tickCount = 0;
    auto lastFpsLogTime = std::chrono::high_resolution_clock::now();

    MSG msg;
    //main loop
    while (true) {
        while (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE)) {
            TranslateMessage(&msg);
            DispatchMessage(&msg);
        
            // Handle WM_QUIT properly
            if (msg.message == WM_QUIT) {
                //memory cleanup
                WinLib_DestroyWindow(window);
                bool endKernel = true;
                hipMemcpyAsync(&gpuMetaData->shouldEndKernel, &endKernel, sizeof(bool), hipMemcpyHostToDevice, stream);
                hipStreamSynchronize(stream);
                hipStreamDestroy(stream);
                hipFree(gpuMetaData);
                hipEventDestroy(interpolatorCopyEvent);
                hipEventDestroy(fpsCopyEvent);
                hipHostFree(displayFrame);
                return 0;
            }
        }

        auto now = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> timeSinceTick = now-lastTickTime;
        std::chrono::duration<float> timeSinceDisplay = now-lastDisplayTime;
        std::chrono::duration<float> timeSinceFpsLog = now - lastFpsLogTime;

        if (timeSinceFpsLog.count() >= 1.0f) {
            int framesBefore = frameCounter;
            hipMemcpyAsync( &frameCounter,&gpuMetaData->framesCalculated,sizeof(int),hipMemcpyDeviceToHost);

            hipEventRecord(interpolatorCopyEvent);
            hipEventSynchronize(interpolatorCopyEvent); 
            char buffer[64];
            sprintf(buffer, "GPU FPS: %d\n", frameCounter-framesBefore);
            OutputDebugString(buffer);

            // Reset counter and update time;
            lastFpsLogTime = now;
        }
        
        //if it is time for a new tick, than run it
        if (timeSinceTick.count() > 1.0f/targetTPS) {
            //game logic

            tickCount++;
            //update interpolator
            interpolator newInterpolator;
            newInterpolator.tickCount = tickCount;

            // Determine the inactive interpolator slot.
            int inactiveIndex = 1 - currentActiveInterpolator;
            
            // Asynchronously copy the new interpolator data to the inactive slot on the GPU.
            hipMemcpyAsync(&gpuMetaData->interpolators[inactiveIndex], &newInterpolator,sizeof(interpolator),hipMemcpyHostToDevice);

            hipEventRecord(interpolatorCopyEvent);
            hipEventSynchronize(interpolatorCopyEvent);                

            
            // Set the flag so the GPU will switch to the new interpolator on its next frame.
            bool switchFlag = true;
            hipMemcpyAsync(&gpuMetaData->shouldSwitchInterpolator, &switchFlag , sizeof(bool) , hipMemcpyHostToDevice);
            
            hipEventRecord(interpolatorCopyEvent);
            hipEventSynchronize(interpolatorCopyEvent);    
            

            // Update our CPU-side record of the active slot.
            // The GPU will switch to 'inactiveIndex' upon processing the flag.
            currentActiveInterpolator = inactiveIndex;

            if (tickCount == 1) { // start up the frame calculations after the first interpolator is made
                frameComputeLoop<<<1,1,0,stream>>>(gpuMetaData,width,height,stream);
            }
            lastTickTime = now;
        }

        //if it is time for a new image to be displayed, do so
        if (timeSinceDisplay.count() > 1.0f / refreshRate) {

            //flag GPU to copy
            bool copyFlag = true;
            hipMemcpyAsync(&gpuMetaData->shouldCopyFrame,&copyFlag,sizeof(bool),hipMemcpyHostToDevice);

            InvalidateRect(window->hwnd,NULL,FALSE);


            lastDisplayTime = now;
        }
    }
    return 0;
}