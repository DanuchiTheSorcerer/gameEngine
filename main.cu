#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <windows.h>
#include <chrono>
#include <climits>
#include "winlib.h"

int width = 1000;
int height = 1000;
float targetTPS = 60;
int refreshRate;
BITMAPINFO bmi;
int activeDisplayBufferIndex = 0;
int currentActiveInterpolator = 0; //cpu side
hipEvent_t frameCopyEvent;

struct frameBuffer {
    int state = 0; // 0 is untouched  1 is being worked on   2 is being copied (or has been copied if on DRAM)
    uint32_t* pixels; // pointer to pixels. Will be assigned at GPU allocation
};

struct interpolator {
    //fully customisable by the person using the engine. Contains all the information needed to compute a frame, minus the interpolation index
    int tickCount; // example data
};

struct gpuMeta {
    frameBuffer buffers[3]; //array of buffers
    int bufferRecencyOrder[3];   // rank of buffer indicies
    interpolator interpolators[2]; // array of interpolators
    int activeInterpolator = 0; // index of the current interpolator being used

    //flags, set only by the CPU and interpreted by GPU when it has the chance
    bool shouldSwitchInterpolator = false;  // for switching interpolator, set after new one has been copied
    bool shouldEndKernel = false; // for ending the persistent kernel
};

gpuMeta* gpuMetaData; // data on the GPU
frameBuffer displayBuffers[3]; //pined memory buffers on DRAM for displaying

void initAll(HINSTANCE hInstance) {
    //intiate winlib
    WinLib_Init(hInstance);

    //get refresh rate
    DEVMODE dm = {0};
    dm.dmSize = sizeof(DEVMODE);
    EnumDisplaySettings(nullptr, ENUM_CURRENT_SETTINGS, &dm);
    refreshRate = dm.dmDisplayFrequency;

    //configure bit map
    ZeroMemory(&bmi, sizeof(bmi));
    bmi.bmiHeader.biSize = sizeof(BITMAPINFOHEADER);
    bmi.bmiHeader.biWidth = width;
    bmi.bmiHeader.biHeight = -height;
    bmi.bmiHeader.biPlanes = 1;
    bmi.bmiHeader.biBitCount = 32;
    bmi.bmiHeader.biCompression = BI_RGB;

    //cuda memory allocations
    hipMalloc(&gpuMetaData,sizeof(gpuMeta));   // allocate GPU data
    gpuMeta initialData = {};  

    for (int i = 0; i<3;i++) {
        hipMalloc(&initialData.buffers[i].pixels, width * height * sizeof(uint32_t));  //allocate buffers 
        hipHostAlloc(&displayBuffers[i].pixels, width * height * sizeof(uint32_t), hipHostMallocDefault); //allocate display buffers to pinned memory for quick copies
    }

    hipMemcpy(gpuMetaData,&initialData,sizeof(gpuMeta), hipMemcpyHostToDevice);

    hipEventCreate(&frameCopyEvent);
}

void PaintWindow(HDC hdc) {
    //draw (this is called by winlib when handling an invalidated rect)
    StretchDIBits(hdc, 0, 0, width, height, 0, 0, width, height,displayBuffers[activeDisplayBufferIndex].pixels, &bmi,DIB_RGB_COLORS, SRCCOPY);
}

__global__ void computeFrame(uint32_t* buffer, interpolator* interpolator,int width,int height) { //pointer to buffer
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    const int idx = y * width + x;

    buffer[idx] = 0xFF000000 | 
        ((x + interpolator->tickCount) % 255) << 16 |
        ((y + interpolator->tickCount) % 255) << 8 |
        ((x + y + interpolator->tickCount) % 255);
}

__global__ void frameComputeLoop(gpuMeta* gpuMetaData, int width, int height,hipStream_t stream) {
    while(!gpuMetaData->shouldEndKernel) {
        //check for new interpolators

        if (gpuMetaData->shouldSwitchInterpolator) {
            gpuMetaData->activeInterpolator = 1 - gpuMetaData->activeInterpolator;
            gpuMetaData->shouldSwitchInterpolator = false;
        }

        // compute a buffer

        dim3 threadsPerBlock(16, 16);     
        dim3 numBlocks((width + 15) / 16, (height + 15) / 16);

        int updatedBufferIndex;

        if (gpuMetaData->buffers[gpuMetaData->bufferRecencyOrder[2]].state != 2) {  // compute on the most outdated buffer if it is not being copied

            updatedBufferIndex = gpuMetaData->bufferRecencyOrder[2];
            gpuMetaData->buffers[updatedBufferIndex].state = 1;
            computeFrame<<<numBlocks,threadsPerBlock>>>(gpuMetaData->buffers[updatedBufferIndex].pixels,&gpuMetaData->interpolators[gpuMetaData->activeInterpolator],width,height);

            gpuMetaData->buffers[updatedBufferIndex].state = 0;

        } else { // if most outdated buffer is being copied, then use the second most outdated

            updatedBufferIndex = gpuMetaData->bufferRecencyOrder[1];
            gpuMetaData->buffers[updatedBufferIndex].state = 1;
            computeFrame<<<numBlocks,threadsPerBlock>>>(gpuMetaData->buffers[updatedBufferIndex].pixels,&gpuMetaData->interpolators[gpuMetaData->activeInterpolator],width,height);
            gpuMetaData->buffers[updatedBufferIndex].state = 0;

        }

        // update recency ordering

        int newOrdering[3];
        newOrdering[0] = updatedBufferIndex;
        newOrdering[1] = gpuMetaData->bufferRecencyOrder[0];
        newOrdering[2] = gpuMetaData->bufferRecencyOrder[updatedBufferIndex % 2 + 1];

        gpuMetaData->bufferRecencyOrder[0] = newOrdering[0];
        gpuMetaData->bufferRecencyOrder[1] = newOrdering[1];
        gpuMetaData->bufferRecencyOrder[2] = newOrdering[2];


    }
}

int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpCmdLine, int nCmdShow) {
    initAll(hInstance); //initialize
    WinWindow* window = WinLib_CreateWindow("CUDA Powered Engine", width, height, hInstance); //open the window

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);



    auto lastTickTime = std::chrono::high_resolution_clock::now();
    auto lastDisplayTime = std::chrono::high_resolution_clock::now();
    int tickCount = 0;
    MSG msg;
    //main loop
    while (GetMessage(&msg,NULL,0,0)) {
        OutputDebugString("Hello\n");
        TranslateMessage(&msg);
        DispatchMessage(&msg);

        auto now = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> timeSinceTick = now-lastTickTime;
        std::chrono::duration<float> timeSinceDisplay = now-lastDisplayTime;
        
        //if it is time for a new tick, than run it
        if (timeSinceTick.count() > 1.0f/targetTPS) {
            OutputDebugString("Logician\n");
            //game logic

            tickCount++;
            //update interpolator
            interpolator newInterpolator;
            newInterpolator.tickCount = tickCount;

            // Determine the inactive interpolator slot.
            int inactiveIndex = 1 - currentActiveInterpolator;
            
            // Asynchronously copy the new interpolator data to the inactive slot on the GPU.
            hipMemcpyAsync(&gpuMetaData->interpolators[inactiveIndex], &newInterpolator,sizeof(interpolator),hipMemcpyHostToDevice,stream);
            
            // Set the flag so the GPU will switch to the new interpolator on its next frame.
            bool switchFlag = true;
            hipMemcpyAsync(&gpuMetaData->shouldSwitchInterpolator, &switchFlag , sizeof(bool) , hipMemcpyHostToDevice, stream);
            
            hipStreamSynchronize(stream);                
            // Update our CPU-side record of the active slot.
            // The GPU will switch to 'inactiveIndex' upon processing the flag.
            currentActiveInterpolator = inactiveIndex;

            if (tickCount == 1) { // start up the frame calculations after the first interpolator is made
                frameComputeLoop<<<1,1,0,stream>>>(gpuMetaData,width,height,stream);
            }
            lastTickTime = now;
        }

        //if it is time for a new image to be displayed, do so
        if (timeSinceDisplay.count() > 1.0f / refreshRate) {
            OutputDebugString("Display\n");
            lastDisplayTime = now;
        
            // Determine target display buffer
            int targetBuffer = (activeDisplayBufferIndex + 1) % 3;
        
            // Get the latest buffer index from device
            int latestBufferIndex;
            hipMemcpyAsync(&latestBufferIndex, &gpuMetaData->bufferRecencyOrder[0], sizeof(int), hipMemcpyDeviceToHost, stream);
            OutputDebugString("Hello chester?\n");
            
            // Ensure latestBufferIndex is ready
            hipStreamSynchronize(stream);
            OutputDebugString("nah\n");
        
            // Copy from device to host asynchronously
            uint32_t** src = &gpuMetaData->buffers[latestBufferIndex].pixels;
            OutputDebugString("Hewwwwwwlo\n");
            uint32_t** dst = &displayBuffers[targetBuffer].pixels;
            OutputDebugString("cwinge\n");

            hipMemcpyAsync(dst, src, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost, stream);
        
            // Record event after copy
            hipEventRecord(frameCopyEvent, stream);
            OutputDebugString("whoat\n");
        
            // Launch callback when copy is done
            hipLaunchHostFunc(stream, [](void* data) {
                WinWindow* window = static_cast<WinWindow*>(data);
                activeDisplayBufferIndex = (activeDisplayBufferIndex + 1) % 3;
                InvalidateRect(window->hwnd, NULL, FALSE);
            }, window);
        }
    }

    //memory cleanup
    WinLib_DestroyWindow(window);
    bool endKernel = true;
    hipMemcpyAsync(&gpuMetaData->shouldEndKernel, &endKernel, sizeof(bool), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    hipFree(gpuMetaData);
    hipEventDestroy(frameCopyEvent);
    return 0;
}