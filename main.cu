#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <windows.h>
#include "winlib.h"
#include <chrono>
#include <string>

const int WIDTH = 800;
const int HEIGHT = 600;
const int BUFFER_COUNT = 3;

uint32_t* gpu_buffers[BUFFER_COUNT];  // Framebuffers on GPU
uint32_t* cpu_buffers[BUFFER_COUNT];  // CPU-accessible buffers
hipEvent_t frameReady[BUFFER_COUNT]; // CUDA events for sync
int currentBuffer = 0;  // Tracks the active buffer

BITMAPINFO bmi;
HDC hdcMem;

// CUDA kernel to fill the framebuffer
__global__ void renderKernel(uint32_t* framebuffer, int width, int height, int frameCount) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;
        framebuffer[index] = 0xFF000000 | ((x + frameCount) % 255) << 16 | ((y + frameCount) % 255) << 8 | 255;
    }
}

// Function to initialize GPU memory
void InitCUDA() {
    for (int i = 0; i < BUFFER_COUNT; i++) {
        hipMalloc(&gpu_buffers[i], WIDTH * HEIGHT * sizeof(uint32_t));
        hipHostMalloc(&cpu_buffers[i], WIDTH * HEIGHT * sizeof(uint32_t));  // Pinned memory
        hipEventCreate(&frameReady[i]);
    }
}

// Function to render a frame
void RenderFrame(int frameCount) {
    int nextBuffer = (currentBuffer + 1) % BUFFER_COUNT;

    // Launch CUDA kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + 15) / 16, (HEIGHT + 15) / 16);
    renderKernel<<<numBlocks, threadsPerBlock>>>(gpu_buffers[nextBuffer], WIDTH, HEIGHT, frameCount);
    hipEventRecord(frameReady[nextBuffer]);  // Signal completion

    // Wait for previous frame to be ready before copying
    hipEventSynchronize(frameReady[currentBuffer]);
    hipMemcpyAsync(cpu_buffers[currentBuffer], gpu_buffers[currentBuffer], WIDTH * HEIGHT * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Invalidate the window to trigger WM_PAINT
}

// Function to display the framebuffer in WM_PAINT
void PaintWindow(HDC hdc) {
    StretchDIBits(hdc, 0, 0, WIDTH, HEIGHT, 0, 0, WIDTH, HEIGHT, cpu_buffers[currentBuffer], &bmi, DIB_RGB_COLORS, SRCCOPY);
    currentBuffer = (currentBuffer + 1) % BUFFER_COUNT;  // Swap buffers
}

// Win32 message loop with rendering
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpCmdLine, int nCmdShow) {
    // Initialize window using winlib
    WinLib_Init(hInstance);
    WinWindow* myWindow = WinLib_CreateWindow("My Win32 Window", WIDTH, HEIGHT, hInstance);

    // Initialize framebuffer
    InitCUDA();

    // Setup DIB
    ZeroMemory(&bmi, sizeof(bmi));
    bmi.bmiHeader.biSize = sizeof(BITMAPINFOHEADER);
    bmi.bmiHeader.biWidth = WIDTH;
    bmi.bmiHeader.biHeight = -HEIGHT;  // Negative to flip vertically
    bmi.bmiHeader.biPlanes = 1;
    bmi.bmiHeader.biBitCount = 32;
    bmi.bmiHeader.biCompression = BI_RGB;

    HDC hdc = GetDC(myWindow->hwnd);
    hdcMem = CreateCompatibleDC(hdc);
    ReleaseDC(myWindow->hwnd, hdc);

    int frameCount = 0;
    int frameCheck = 0;
    auto lastFrameTime = std::chrono::steady_clock::now();
    MSG msg;
    while (GetMessage(&msg, NULL, 0, 0)) {
        // Process Windows messages
        TranslateMessage(&msg);
        DispatchMessage(&msg);

        // Calculate time since last frame
        auto currentFrameTime = std::chrono::steady_clock::now();
        std::chrono::duration<float> deltaTime = currentFrameTime - lastFrameTime;
        if (deltaTime.count() > 1) {
            std::string debugString = std::to_string(frameCheck) + "fps\n";
            OutputDebugString(debugString.c_str());
            frameCheck = 0;
            lastFrameTime = currentFrameTime;
        }
        // Render frame
        RenderFrame(frameCount++);
        frameCheck++;

        // Request a repaint AFTER processing events
        InvalidateRect(myWindow->hwnd, NULL, FALSE);


    }


    // Cleanup
    for (int i = 0; i < BUFFER_COUNT; i++) {
        hipFree(gpu_buffers[i]);
        hipHostFree(cpu_buffers[i]);
        hipEventDestroy(frameReady[i]);
    }

    // Destroy window using winlib
    WinLib_DestroyWindow(myWindow);
    return 0;
}
